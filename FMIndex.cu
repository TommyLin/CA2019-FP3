#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include <sys/time.h>

using namespace std;
int **L_counts;
int compSuffixes(char *suffix1, char *suffix2, int length);

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Value that will be compared for correctness
//You need to create the function prototypes and definitions as per your design, but you need to present final results in this array
//-----------------------------Structures for correctness check-------------------
char **fourbit_sorted_suffixes_student;
int read_count = 0;
int read_length = 0;
int num_value = 0;
char **fourbit_sorted_suffixes_original;
int BLOCKS, THREADS;
char* fourbitEncodeRead(char *read, int length);
char** generateSuffixes(char *read, int byte_length);
char ctable[] = {'$', 'A', 'C', 'G', 'T', '5', '6', '7',
	             '8', '9', 'A', 'B', 'C', 'D', 'E', 'F'};

char **student;

void print_string_2d(char **str, int len , int cnt){
	printf("=== string address ===\n");
	printf("\n");
	for (int i = 0; i < 2 * len; i++) {
		printf(" %X", i);
	}
	printf("\n============== 2d print ==============\n");
	printf("============== read_count = %d ==============\n",read_count);

	for (int i = 0; i < len*cnt; i++) {
		for (int z = 0; z < len/2; z++){
			printf("%c%c", ctable[str[i][z]>>4], ctable[str[i][z] &0xF]);
			//fout<<ctable[str[i][z]>>4]<<ctable[str[i][z] &0xF];
		}
		printf("\n");
		//fout<<"\n";
	}
}
void print_string_1d(char *str, int len){
	printf("=== string address ===\n");
	cout<<"input string is "<<str<<endl;
	cout<<"lengh = "<<len<<endl;
	cout<<"num_value = "<<num_value<<endl;

	printf("================================\n");
	for (int i = 0; i < 2 * len; i++) {
		printf(" %X", i);
	}
	printf("\n============== 1d print ==============\n");

	for (int i = 0; i < num_value; i++) {
		for (int z = 0; z < len/2; z++){
			printf("%c%c", ctable[str[(i*len/2+z)]>>4 ], ctable[str[i*len/2+z] &0xF]);
		}
		printf("\n");
	}
	printf("\n============== 1d print ==============\n");
}
__global__ void fourbitEncodeRead_gpu(char *dev_read, int length, int i){

	char this_char = dev_read[i];
	char fourbit_char;
	if(this_char == '$')
		fourbit_char = 0x00;
	else if(this_char == 'A')
		fourbit_char = 0x01;
	else if(this_char == 'C')
		fourbit_char = 0x02;
	else if(this_char == 'G')
		fourbit_char = 0x03;
	else
		fourbit_char = 0x04;
	fourbit_char = i%2==0 ? fourbit_char << 4 : fourbit_char;
	dev_read[i/2] = dev_read[i/2] | fourbit_char;

}

char* fourbitEncodeRead_stu(char *read, int length){
    int byte_length = length/2;
    char *fourbit_read = (char*)calloc(byte_length,sizeof(char));
	char *dev_read;
	hipMalloc((void**) &dev_read, length*sizeof(char));
	hipMemcpy(dev_read, read, length*sizeof(char), hipMemcpyHostToDevice);
	dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */
    for(int i=0;i<length;i++){
        fourbitEncodeRead_gpu <<<blocks, threads>>> (dev_read, length , i);
    }
	hipMemcpy(fourbit_read, dev_read, byte_length*sizeof(char) , hipMemcpyDeviceToHost);
	hipFree(dev_read);
   return fourbit_read;
}

__global__ void rotateRead_gpu_part1(char *dev_read, int i , char prev_4bit){
	char this_char = ((dev_read[i] >> 4) & 0x0F) | prev_4bit;
	dev_read[i] = this_char;
}


char* rotateRead_stu(char *read, int byte_length){

    char prev_4bit = (read[0] & 0x0F) << 4;
	char *dev_read;
	dim3 blocks(BLOCKS,1);
    dim3 threads(THREADS,1);
	hipMalloc((void**) &dev_read, byte_length*sizeof(char));
	hipMemcpy(dev_read, read, byte_length*sizeof(char), hipMemcpyHostToDevice);
    for(int i=1;i<byte_length;i++){
		rotateRead_gpu_part1 <<<blocks, threads>>> (dev_read , i , (read[i-1] & 0x0F) << 4);
    }
	prev_4bit = (read[byte_length-1] & 0x0F) << 4;
	hipMemcpy(read, dev_read, byte_length*sizeof(char) , hipMemcpyDeviceToHost);
	hipFree(dev_read);
	read[0] = (read[0] >> 4) & 0x0F;
    read[0]=read[0] | prev_4bit;

    char *rotated_read = (char*)malloc(byte_length*sizeof(char));

    for(int i=0;i<byte_length;i++){
        rotated_read[i] = read[i];
	}

    return rotated_read;
}
/*
char* rotateRead_stu(char *read, int byte_length){

    char prev_4bit = (read[0] & 0x0F) << 4;
    read[0] = (read[0] >> 4) & 0x0F;
    for(int i=1;i<byte_length;i++){
        char this_char = ((read[i] >> 4) & 0x0F) | prev_4bit;
        prev_4bit = (read[i] & 0x0F) << 4;
        read[i] = this_char;
    }
    read[0]=read[0] | prev_4bit;
    char *rotated_read = (char*)malloc(byte_length*sizeof(char));
    for(int i=0;i<byte_length;i++)
        rotated_read[i] = read[i];
    return rotated_read;
}*/
//Generate Sufixes for a 4-bit encoded read
char** generateSuffixes_stu(char *read, int byte_length){

	fourbitEncodeRead_stu(read, read_length);

    char **suffixes=(char**)malloc(byte_length*2*sizeof(char*));
    for(int i=0;i<byte_length*2;i++){
        suffixes[i] = rotateRead_stu(read, byte_length);
    }
    return suffixes;
}



__global__ void bitonic_sort_step(char *dev_values, int j, int k, int num_value, int read_length, int read_count){
    //printf(">>> bitonic_sort_step\n");
    int flag = 0;
	int HIGH = 0;
	unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;
    char temp_char_i,temp_char_ixj;
	//printf("input string = %s\n",dev_values);
    /* The threads with the lowest ids sort the array. */
    flag = 0;
    if ((ixj)>i) {
        for(int l=0;l<read_length;l++){
			if (HIGH) {
				temp_char_i   = dev_values[i  *read_length / 2 + l / 2] & (0xF);
				temp_char_ixj = dev_values[ixj*read_length / 2 + l / 2] & (0xF);
			} else {
				temp_char_i   = (dev_values[i  *read_length / 2 + l / 2]& (0xF0)) >>4;
				temp_char_ixj = (dev_values[ixj*read_length / 2 + l / 2]& (0xF0)) >>4;
			}

			if (temp_char_i>temp_char_ixj){
				flag = 1;
                break;
            } else if(temp_char_i<temp_char_ixj){
                flag = -1;
                break;
            }
            HIGH = !HIGH;

        }
        //printf("i=%d, ixj=%d, sorting result flag = %d\n",i,ixj,flag);


        if ((i&k)==0) {
            // Sort ascending //
            if (flag==1) {
                char* temp;
				temp = (char*)malloc(sizeof(char)*read_length/2);
				memcpy(temp, &dev_values[i*read_length/2], read_length/2*sizeof(char));
				memcpy(&dev_values[i*read_length/2], &dev_values[ixj*read_length/2], read_length/2*sizeof(char));
				memcpy(&dev_values[ixj*read_length/2], temp, read_length/2*sizeof(char));
				free(temp);
            }
        }
        if ((i&k)!=0) {
            // Sort descending

            if (flag==-1) {
				char* temp;
				temp = (char*)malloc(sizeof(char)*read_length/2);
				memcpy(temp, &dev_values[i*read_length/2], read_length/2*sizeof(char));
				memcpy(&dev_values[i*read_length/2], &dev_values[ixj*read_length/2], read_length/2*sizeof(char));
				memcpy(&dev_values[ixj*read_length/2], temp, read_length/2*sizeof(char));
				free(temp);
            }
        }
    }
}
void bitonic_sort(char **values, fstream& fout){
    char *dev_values;
    size_t size = read_length/2 * sizeof(char);
    char *temp;
    char *temp_char = new char[read_length/2];


    temp = (char*)malloc(num_value*size);
    for(int i=0;i<read_length/2;i++){
        temp_char[i]=0x44;
    }
	for (int i = 0; i < num_value; i++){
        if (i < read_length * read_count){
            memcpy(&temp[i*read_length/2], values[i], size);
        }
        else{
            memcpy(&temp[i*read_length/2], temp_char , 	size);
        }
    }
	free(temp_char);
	//printf("001\n");
    hipMalloc((void**) &dev_values, size*num_value);

    hipMemcpy(dev_values, temp, num_value*size, hipMemcpyHostToDevice);
	//cout<<"================debug======================"<<endl;
	//print_string_1d(temp,read_length);
    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */
	//cout<<"=========== before temp ==========="<<endl;
	//print_string_1d (temp,read_length);
	//cout<<"=========== after temp ==========="<<endl;
    int j, k;
    /* Major step */

    for (k = 2; k <= num_value; k <<= 1) {
        //* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
			bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k, num_value,read_length, read_count);
			//bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k, num_value,read_length, 1);
		}
    }

    hipMemcpy(temp, dev_values, read_length*read_count*size, hipMemcpyDeviceToHost);
    //hipMemcpy(temp, dev_values, read_length*1*size, hipMemcpyDeviceToHost);

	for(int i=0;i<read_length*read_count;i++){

        memcpy(values[i],&temp[i*read_length/2],read_length/2*sizeof(char));

    }


    /*for(int i=0;i<num_value;i++){
        if(i<read_length*read_count){
            memcpy(values[i],&temp[i*read_length],read_length*sizeof(char));
        }
        else{
            memcpy(temp_char,&temp[i*read_length],read_length*sizeof(char));
        }
    }	*/
	//print_string_2d(fourbit_sorted_suffixes_student, read_length,read_count);
    //cout<<"begin teeeeeeeeeeeeeeeeeeeeeeeeeeeeeemp"<<endl;


	free(temp);
    hipFree(dev_values);
}


void pipeline_stu(char **reads, int read_length, int read_count, fstream& fout){
	int temp_stu = ceil(log2((float)read_length*read_count));

	num_value = pow(2,temp_stu);
	if(num_value<=256){
		THREADS = num_value;
		BLOCKS = 1;
	}
	else{
		THREADS = 256;
		BLOCKS = num_value/THREADS;
	}
    fourbit_sorted_suffixes_student = (char**)malloc(read_length*read_count*sizeof(char*));

    for(int i=0;i<read_count;i++){
        char **suffixes_for_read = generateSuffixes(fourbitEncodeRead(reads[i], read_length), read_length/2);
		//cout << "read_length = " << read_length << endl;

		//bitonic_sort(suffixes_for_read);

        for(int j=0;j<read_length;j++){
            fourbit_sorted_suffixes_student[i*read_length+j] = suffixes_for_read[j];
        }
		free(suffixes_for_read);
    }
	cout<<"=========== before bitonic_sort ==========="<<endl;
	//print_string_2d(fourbit_sorted_suffixes_student, read_length,read_count, fout);
	cout<<"=========== into bitonic_sort ==========="<<endl;
	bitonic_sort(fourbit_sorted_suffixes_student, fout);


    //--------------For debug purpose--------------
    /*
    for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++)
            printf("%x\t",fourbit_sorted_suffixes_original[i][j]);
        printf("\n");
    }*/
    //---------------------------------------------
}

//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE AT ALL--------------------------------------------
int **SA_Final;
char *L;
int F_counts[]={0,0,0,0};

//This array is the default result



//Read file to get reads
char** inputReads(const char *file_path, int *read_count, int *length){//same
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do
    {
        ch = fgetc(read_file);
        if (ch == '\n')
            lines++;
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;
    size_t len = 0;
    for(i = 0; i < lines; i++)
    {
        reads[i] = NULL;
        len = 0;
        getline(&reads[i], &len, read_file);
    }
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
    return reads;
}


//Check correctness of values
int checker(){
    int correct = 1;
	//print_string_2d(fourbit_sorted_suffixes_student, read_length,read_count);
    //print_string_2d(fourbit_sorted_suffixes_original, read_length,read_count);

	for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++){
            if(fourbit_sorted_suffixes_student[i][j] != fourbit_sorted_suffixes_original[i][j]){
				correct = 0;
				/*cout<<"wrong i="<<i<<" wrong j = "<<j<<endl;
				print_string_1d(fourbit_sorted_suffixes_student[i],read_length);*/
				//print_string_1d(fourbit_sorted_suffixes_original[i],read_length);
			}
        }
    }
    return correct;
}

//Rotate 4-bit encoded read by 1 character (4-bit)
char* rotateRead(char *read, int byte_length){//rotateRead_2
    char prev_4bit = (read[0] & 0x0F) << 4;
    read[0] = (read[0] >> 4) & 0x0F;
    for(int i=1;i<byte_length;i++){
        char this_char = ((read[i] >> 4) & 0x0F) | prev_4bit;
        prev_4bit = (read[i] & 0x0F) << 4;
        read[i] = this_char;
    }
    read[0]=read[0] | prev_4bit;
    char *rotated_read = (char*)malloc(byte_length*sizeof(char));
    for(int i=0;i<byte_length;i++)
        rotated_read[i] = read[i];
    return rotated_read;
}
void rotateRead_2(char *read, char *rotatedRead, int length){//2
    for(int i=0;i<length-1;i++)
        rotatedRead[i]=read[i+1];
    rotatedRead[length-1]=read[0];
}


//Generate Sufixes for a 4-bit encoded read
char** generateSuffixes(char *read, int byte_length){//generateSuffixes_2
    char **suffixes=(char**)malloc(byte_length*2*sizeof(char*));
    for(int i=0;i<byte_length*2;i++){
        suffixes[i] = rotateRead(read, byte_length);
    }
    return suffixes;
}
char** generateSuffixes_2(char *read, int length, int read_id){//2
    char **suffixes=(char**)malloc(length*sizeof(char*));
    suffixes[0]=(char*)malloc(length*sizeof(char));
    for(int j=0;j<length;j++)
        suffixes[0][j]=read[j];
    for(int i=1;i<length;i++){
        suffixes[i]=(char*)malloc(length*sizeof(char));
        rotateRead_2(suffixes[i-1], suffixes[i], length);
    }
    return suffixes;
}

//Comparator for 4-bit encoded Suffixes
int compSuffixes(char *suffix1, char *suffix2, int byte_length){//same
    int ret = 0;
    for(int i=0;i<byte_length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}

char* fourbitEncodeRead(char *read, int length){
    int byte_length = length/2;
    char *fourbit_read = (char*)calloc(byte_length,sizeof(char));
    for(int i=0;i<length;i++){
        char this_char = read[i];
        char fourbit_char;
        if(this_char == '$')
            fourbit_char = 0x00;
        else if(this_char == 'A')
            fourbit_char = 0x01;
        else if(this_char == 'C')
            fourbit_char = 0x02;
        else if(this_char == 'G')
            fourbit_char = 0x03;
        else
            fourbit_char = 0x04;
        fourbit_char = i%2==0 ? fourbit_char << 4 : fourbit_char;
        fourbit_read[i/2] = fourbit_read[i/2] | fourbit_char;
    }
   return fourbit_read;
}

void sort_fourbit_suffixes(char **suffixes, int suffix_count, int byte_length){
    char *temp=(char*)malloc(byte_length*sizeof(char));
    for(int i=0;i<suffix_count-1;i++){
        for(int j=0;j<suffix_count-i-1;j++){
            if(compSuffixes(suffixes[j], suffixes[j+1], byte_length)>0){
                memcpy(temp, suffixes[j], byte_length*sizeof(char));
                memcpy(suffixes[j], suffixes[j+1], byte_length*sizeof(char));
                memcpy(suffixes[j+1], temp, byte_length*sizeof(char));
            }
        }
    }
	free(temp);
}

int** makeFMIndex(char ***suffixes, int read_count, int read_length, int F_count[], char *L){//2
    int i, j;

    SA_Final=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(read_length*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final[i*read_length+j][0]=j;
            SA_Final[i*read_length+j][1]=i;
        }
    }

    char *temp=(char*)malloc(read_length*sizeof(char));

    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }


    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final[j][0];
                SA_Final[j][0]=SA_Final[j+1][0];
                SA_Final[j+1][0]=temp_int;
                temp_int = SA_Final[j][1];
                SA_Final[j][1]=SA_Final[j+1][1];
                SA_Final[j+1][1]=temp_int;
            }
        }
    }

    free(temp);
    char this_F = '$';
    j=0;

    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }

    //Calculation of L's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }
	//for(int i=0; i<read_count*read_length; ++i) free(temp_suffixes[i]);
	free(temp_suffixes);
    return L_count;
}

//Default Pipeline. You need to implement CUDA function corresponding to everything inside this function
void pipeline(char **reads, int read_length, int read_count){
    fourbit_sorted_suffixes_original = (char**)malloc(read_length*read_count*sizeof(char*));
    for(int i=0;i<read_count;i++){
        char **suffixes_for_read = generateSuffixes(fourbitEncodeRead(reads[i], read_length), read_length/2);
        sort_fourbit_suffixes(suffixes_for_read, read_length, read_length/2);

        for(int j=0;j<read_length;j++){
            fourbit_sorted_suffixes_original[i*read_length+j] = suffixes_for_read[j];
        }
    }
    //--------------For debug purpose--------------
    /*
    for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++)
            printf("%x\t",fourbit_sorted_suffixes_original[i][j]);
        printf("\n");
    }*/
    //---------------------------------------------
}

// void Merge(char** suffixes, int front, int mid, int end){
	// char** LeftSub = (char**) malloc((mid-front+1+1)*sizeof(char*));
	// char** RightSub = (char**) malloc((end-mid+1)*sizeof(char*));
	// char* MAXchar = (char*) malloc(read_length/2*sizeof(char));
	// for(int i=0; i<read_length/2; ++i)
		// MAXchar[i] = 0x44;
	// memcpy(LeftSub[mid-front+1+1-1], MAXchar, sizeof(char*));
	// memcpy(LeftSub[end-mid+1-1], MAXchar, sizeof(char*));
	// memcpy(LeftSub, &suffixes[front], (mid-front+1)*sizeof(char*));
	// memcpy(LeftSub, &suffixes[front], (mid-front+1)*sizeof(char*));

    // int idxLeft = 0, idxRight = 0;

    // for (int i = front; i <= end; i++) {

        // if (LeftSub[idxLeft] <= RightSub[idxRight] ) {
            // Array[i] = LeftSub[idxLeft];
            // idxLeft++;
        // }
        // else{
            // Array[i] = RightSub[idxRight];
            // idxRight++;
        // }
    // }
// }
//Merge all sorted suffixes in overall sorted order
// void mergeAllSorted4bitSuffixes(char** suffixes, int read_count, int read_length){
	// int flag = 0;
	// int HIGH = 0;
    // char temp_char_i,temp_char_j;
	// for(int i=0;i<read_count;i++)
		// for(int j=0;j<read_length;j++){
			// for(int k=i+j*read_length;k<read_count*read_length;k++){
				// for(int l=0;l<read_length;l++){
					// if (HIGH)        temp_char_i   = suffixes[i*read_length/2+l/2]&(0xF);
					// else if(!HIGH)    temp_char_i   = (suffixes[i*read_length/2+l/2]&(0xF0))>>4;
					// if (HIGH)      temp_char_j = suffixes[j*read_length/2+l/2]&(0xF);
					// else if (!HIGH) temp_char_j = (suffixes[j*read_length/2+l/2]&(0xF0))>>4;
					// if(temp_char_i>temp_char_j){
						// flag = 1;
						// break;
					// }
					// else if(temp_char_i<temp_char_j){
						// flag = -1;
						// break;
					// }
					// HIGH = !HIGH;
					// flag = 0;
				// }
			// }
			// if()
		// }
// }

//-----------------------DO NOT CHANGE--------------------------------------------


int main(int argc, char *argv[]){
	char **reads;
	cout << "argc\t= " << argc <<endl;
	cout << "argv[0]\t= " << argv[0] <<endl;

	if (argc > 1) {
		cout << "argv[1]\t= " << argv[1] <<endl;
		reads = inputReads(argv[1], &read_count, &read_length); // Input reads from file
	} else
		reads = inputReads("small.txt", &read_count, &read_length); // Input reads from default file "small.txt"

	cout<<"test00"<<endl;

    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;
	cout<<"test1"<<endl;
    char ***suffixes=(char***)malloc(read_count*sizeof(char**));//Storage for read-wise suffixes
	char **suffixes_encode=(char**)malloc(read_count*read_length*sizeof(char*));
	for(int i=0; i<read_count*read_length; ++i)suffixes_encode[i] = (char*)malloc(read_length/2*sizeof(char));
	cout<<"test2"<<endl;
    L=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
	cout<<"test3"<<endl;
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    // pipeline(reads, read_length, read_count);
    // mergeAllSorted4bitSuffixes(fourbit_sorted_suffixes_original, read_count, read_length);
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes_2(reads[i], read_length, i);
        //suffixes[i]=generateSuffixes(reads[i], read_length);
    }
    L_counts = makeFMIndex(suffixes, read_count, read_length, F_counts, L);
	free(L_counts);
	cout<<"test4-------------------------------------------------------------------"<<endl;

	fstream fout;
	fout.open("s1.txt", ios::out);
	for(int i=0; i<read_count; ++i){
		for(int j=0; j<read_length; ++j){
			//fprintf(stderr,"==============debug=========== %d %d \n",i , j);
			memcpy(suffixes_encode[i*read_length+j],fourbitEncodeRead(suffixes[i][j],read_length),read_length/2*sizeof(char));
			//cout<<suffixes[i][j]<<endl;
		}
	}
	fourbit_sorted_suffixes_original = suffixes_encode;
	fout.close();

    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    cout<<time_overhead_default<<endl;
    //------------Time capture end----------------------
    //--------------------------------------------------


    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------
	cout<<"pipeline_stu"<<endl;

	//fout.open("s2.txt", ios::out);
	pipeline_stu(reads, read_length, read_count, fout);
	cout<<"test5"<<endl;
	//fout.close();

    //-----------Call your functions here--------------------
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
	time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    //--------------------------------------------------


    //---------------Correction check and speedup calculation----------------------

    float speedup=0.0;
    if(checker()==1){
		cout<<"checker()==1"<<endl;
	}
    //speedup = time_overhead_default/time_overhead_student;
    speedup = time_overhead_default/time_overhead_student;
	cout<<"Speedup="<<speedup<<endl;
    //-----------------------------------------------------------------------------

    return 0;
}
