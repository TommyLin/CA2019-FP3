#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include <sys/time.h>

using namespace std;

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Value that will be compared for correctness
//You need to create the function prototypes and definitions as per your design, but you need to present final results in this array
//-----------------------------Structures for correctness check-------------------
char **fourbit_sorted_suffixes_student;
int read_count = 0;
int read_length = 0;
int num_value = 0;
char **fourbit_sorted_suffixes_original;
int BLOCKS, THREADS;
char* fourbitEncodeRead(char *read, int length);
char** generateSuffixes(char *read, int byte_length);
char ctable[] = {'$', 'A', 'C', 'G', 'T', '5', '6', '7',
	'8', '9', 'A', 'B', 'C', 'D', 'E', 'F'};
	
char **student;

void print_string_2d(char **str, int len){
	printf("=== string address ===\n");
	for (int i = 0; i < len; i++) {
		for(int z = 0; z < len/2 ; z++){
			printf("%p ", &(str[i][z]));
		}
		printf("\n");
	}
	printf("\n");

	printf("================================\n");
	for (int i = 0; i < 2 * len; i++) {
		printf(" %X", i);
	}
	printf("\n============== 2d print ==============\n");

	for (int i = 0; i < len; i++) {
		for (int z = 0; z < len/2; z++){
			printf(" %c %c", ctable[str[i][z]>>4], ctable[str[i][z] &0xF]);
		}
		printf("\n");
	}
}
void print_string_1d(char *str, int len){
	printf("=== string address ===\n");
	cout<<"input string is "<<str<<endl;
	cout<<"lengh = "<<len<<endl;
	cout<<"num_value = "<<num_value<<endl;

	printf("================================\n");
	for (int i = 0; i < 2 * len; i++) {
		printf(" %X", i);
	}
	printf("\n============== 1d print ==============\n");

	for (int i = 0; i < num_value; i++) {
		for (int z = 0; z < len/2; z++){
			printf(" %c %c", ctable[str[(i*len/2+z)]>>4 ], ctable[str[i*len/2+z] &0xF]);
		}		
		printf("\n");
	}
	printf("\n============== 1d print ==============\n");
}


__global__ void bitonic_sort_step(char *dev_values, int j, int k, int num_value, int read_length, int read_count){
    //printf("gfdgfdgdsfg\n");
    int flag = 0;
	unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;
    char temp_char_i,temp_char_ixj;
	printf("input string = %s\n",dev_values);
    /* The threads with the lowest ids sort the array. */
    flag = 0;
    if ((ixj)>i) {
        for(int l=0;l<read_length;l++){
			printf("lower char for i = %c\n",dev_values[i*read_length/2+l]&(0xF));
			printf("higher char for i = %c\n",(dev_values[i*read_length/2+l]&(0xF0))>>4);
			
			printf("lower char for ixj = %c\n",dev_values[ixj*read_length/2+l]&(0xF));
			printf("higher char for ixj = %c\n",(dev_values[ixj*read_length/2+l]&(0xF0))>>4);
			
			if(i%2==0) temp_char_i = dev_values[i*read_length/2+l]&(0xF);
			else if(i%2==1) temp_char_i = dev_values[i*read_length/2+l]>>4;
			if(ixj%2==0) temp_char_ixj = dev_values[ixj*read_length/2+l]&(0xF);
			else if(ixj%2==1) temp_char_ixj = dev_values[ixj*read_length/2+l]>>4;
            printf("compare data:\n%d\t%c\n%d\t%c\n",i,temp_char_i,ixj,temp_char_ixj);
			if(temp_char_i>temp_char_ixj){
                //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf(">>>>>>>>>>>>>>>>>>\n");
				flag = 1;
                break;
            }
            else if(temp_char_i<temp_char_ixj){
                //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("<<<<<<<<<<<<<<<<<<<<\n");
                flag = -1;
                break;
            }
            //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("=========================\n");
            flag = 0;

        }
        //printf("i=%d, ixj=%d, sorting result flag = %d\n",i,ixj,flag);


        if ((i&k)==0) {
            // Sort ascending //
            //printf("1110");
            //for(int m=0;m<num_value;m++){

                if (flag==1) {
                    //printf("3333, %d, %d\n", i, ixj);
                    char* temp;
					temp = (char*)malloc(sizeof(char)*read_length/2);
					memcpy(temp, &dev_values[i*read_length/2], read_length/2*sizeof(char));
					memcpy(&dev_values[i*read_length/2], &dev_values[ixj*read_length/2], read_length/2*sizeof(char));
					memcpy(&dev_values[ixj*read_length/2], temp, read_length/2*sizeof(char));
					free(temp);
                }
        }
        if ((i&k)!=0) {
            // Sort descending

            if (flag==-1) {
                //printf("2222, %d, %d\n", i, ixj);
                    char* temp;
					temp = (char*)malloc(sizeof(char)*read_length/2);
					memcpy(temp, &dev_values[i*read_length/2], read_length/2*sizeof(char));
					memcpy(&dev_values[i*read_length/2], &dev_values[ixj*read_length/2], read_length/2*sizeof(char));
					memcpy(&dev_values[ixj*read_length/2], temp, read_length/2*sizeof(char));
					free(temp);
            }
        }
    }
}
void bitonic_sort(char **values){
    char *dev_values;
    size_t size = read_length/2 * sizeof(char);
    char *temp;
    char *temp_char = new char[read_length/2];
    temp = (char*)malloc(num_value*size);
    for(int i=0;i<read_length/2;i++){
        temp_char[i]=0x44;
    }
    for(int i=0;i<num_value;i++){
        if(i<read_length*read_count){
            memcpy(&temp[i*read_length/2],values[i],size);
        }
        else{
            memcpy(&temp[i*read_length/2],temp_char,size);
        }
    }
    hipMalloc((void**) &dev_values, size*num_value);

    hipMemcpy(dev_values, temp, num_value*size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */
	cout<<"=========== before temp ==========="<<endl;
	print_string_1d (temp,read_length*read_count);
	cout<<"=========== after temp ==========="<<endl;
    int j, k;
    /* Major step */

    for (k = 2; k <= 2; k <<= 1) {
        //* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
			bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k, num_value,read_length, read_count);
		}
    }
	
    hipMemcpy(temp, dev_values, read_length*read_count*size, hipMemcpyDeviceToHost);
	
	 for(int i=0;i<read_length*read_count;i++){

        memcpy(values[i],&temp[i*read_length/2],read_length/2*sizeof(char));

    }  
	
	
    /*for(int i=0;i<num_value;i++){
        if(i<read_length*read_count){
            memcpy(values[i],&temp[i*read_length],read_length*sizeof(char));
        }
        else{
            memcpy(temp_char,&temp[i*read_length],read_length*sizeof(char));
        }
    }	*/
	print_string_2d(values, read_length);
    //cout<<"begin teeeeeeeeeeeeeeeeeeeeeeeeeeeeeemp"<<endl;


	free(temp);
    hipFree(dev_values);
}


void pipeline_stu(char **reads, int read_length, int read_count){
	int temp_stu = ceil(log2((float)read_length));
	num_value = pow(2,temp_stu);
	if(num_value<=256){
		THREADS = num_value;
		BLOCKS = 1;
	}
	else{
		THREADS = 256;
		BLOCKS = num_value/THREADS;
	}
    fourbit_sorted_suffixes_student = (char**)malloc(read_length*read_count*sizeof(char*));
    for(int i=0;i<read_count;i++){
        char **suffixes_for_read = generateSuffixes(fourbitEncodeRead(reads[i], read_length), read_length/2);
		cout << "read_length = " << read_length << endl;
		for(int z = 0; z < read_length ; z++){
			//char temp = (z%2==0)?(*suffixes_for_read[z]&0x0f):(*suffixes_for_read[z]&0xf0);
		}
		print_string_2d(suffixes_for_read, read_length);
		bitonic_sort(suffixes_for_read);
			//cout<<**suffixes_for_read <<endl;
        //sort_fourbit_suffixes(suffixes_for_read, read_length, read_length/2);
        for(int j=0;j<read_length;j++){
            fourbit_sorted_suffixes_student[i*read_length+j] = suffixes_for_read[j];
        }
    }
	
	

    //--------------For debug purpose--------------
    /*
    for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++)
            printf("%x\t",fourbit_sorted_suffixes_original[i][j]);
        printf("\n");
    }*/
    //---------------------------------------------
}

//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE AT ALL--------------------------------------------



//This array is the default result



//Read file to get reads
char** inputReads(char *file_path, int *read_count, int *length){
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do                                                                                                 
    {                                                                                                  
        ch = fgetc(read_file);                                                                            
        if (ch == '\n')                                                                                
            lines++;                                                                                   
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;                                                                                         
    size_t len = 0;                                                                                    
    for(i = 0; i < lines; i++)                                                                         
    {
        reads[i] = NULL;
        len = 0;                                                                                
        getline(&reads[i], &len, read_file);
    }                                                                                                  
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
    return reads;
}

//Check correctness of values
int checker(){
    int correct = 1;
    for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++){
            if(fourbit_sorted_suffixes_student[i][j] != fourbit_sorted_suffixes_original[i][j])
                correct = 0;
        }
    }
    return correct;
}

//Rotate 4-bit encoded read by 1 character (4-bit)
char* rotateRead(char *read, int byte_length){
    char prev_4bit = (read[0] & 0x0F) << 4;
    read[0] = (read[0] >> 4) & 0x0F;
    for(int i=1;i<byte_length;i++){
        char this_char = ((read[i] >> 4) & 0x0F) | prev_4bit;
        prev_4bit = (read[i] & 0x0F) << 4;
        read[i] = this_char;
    }
    read[0]=read[0] | prev_4bit;
    char *rotated_read = (char*)malloc(byte_length*sizeof(char));
    for(int i=0;i<byte_length;i++)
        rotated_read[i] = read[i];
    return rotated_read;
}


//Generate Sufixes for a 4-bit encoded read
char** generateSuffixes(char *read, int byte_length){
    char **suffixes=(char**)malloc(byte_length*2*sizeof(char*));
    for(int i=0;i<byte_length*2;i++){
        suffixes[i] = rotateRead(read, byte_length);
    }
    return suffixes;
}

//Comparator for 4-bit encoded Suffixes
int compSuffixes(char *suffix1, char *suffix2, int byte_length){
    int ret = 0;
    for(int i=0;i<byte_length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}

char* fourbitEncodeRead(char *read, int length){
    int byte_length = length/2;
    char *fourbit_read = (char*)calloc(byte_length,sizeof(char));
    for(int i=0;i<length;i++){
        char this_char = read[i];
        char fourbit_char;
        if(this_char == '$')
            fourbit_char = 0x00;
        else if(this_char == 'A')
            fourbit_char = 0x01;
        else if(this_char == 'C')
            fourbit_char = 0x02;
        else if(this_char == 'G')
            fourbit_char = 0x03;
        else
            fourbit_char = 0x04;
        fourbit_char = i%2==0 ? fourbit_char << 4 : fourbit_char;
        fourbit_read[i/2] = fourbit_read[i/2] | fourbit_char;
    }
   return fourbit_read;
}

void sort_fourbit_suffixes(char **suffixes, int suffix_count, int byte_length){
    char *temp=(char*)malloc(byte_length*sizeof(char));
    for(int i=0;i<suffix_count-1;i++){
        for(int j=0;j<suffix_count-i-1;j++){
            if(compSuffixes(suffixes[j], suffixes[j+1], byte_length)>0){
                memcpy(temp, suffixes[j], byte_length*sizeof(char));
                memcpy(suffixes[j], suffixes[j+1], byte_length*sizeof(char));
                memcpy(suffixes[j+1], temp, byte_length*sizeof(char));
            }
            
        }
    }
}

//Default Pipeline. You need to implement CUDA function corresponding to everything inside this function
void pipeline(char **reads, int read_length, int read_count){
    fourbit_sorted_suffixes_original = (char**)malloc(read_length*read_count*sizeof(char*));
    for(int i=0;i<read_count;i++){
        char **suffixes_for_read = generateSuffixes(fourbitEncodeRead(reads[i], read_length), read_length/2);
        sort_fourbit_suffixes(suffixes_for_read, read_length, read_length/2);

        for(int j=0;j<read_length;j++){
            fourbit_sorted_suffixes_original[i*read_length+j] = suffixes_for_read[j];
        }
    }
    //--------------For debug purpose--------------
    /*
    for(int i=0;i<read_count*read_length;i++){
        for(int j=0;j<read_length/2;j++)
            printf("%x\t",fourbit_sorted_suffixes_original[i][j]);
        printf("\n");
    }*/
    //---------------------------------------------
}

//Merge all sorted suffixes in overall sorted order
void mergeAllSorted4bitSuffixes(char** suffixes, int read_count, int read_length){

}

//-----------------------DO NOT CHANGE--------------------------------------------

int main(int argc, char *argv[]){
    char **reads = inputReads(argv[1], &read_count, &read_length);//Input reads from file

    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;

    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    pipeline(reads, read_length, read_count);
    mergeAllSorted4bitSuffixes(fourbit_sorted_suffixes_original, read_count, read_length);

    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    cout<<time_overhead_default<<endl;
    //------------Time capture end----------------------
    //--------------------------------------------------


    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------
	cout<<"pipeline_stu"<<endl;
	pipeline_stu(reads, read_length, read_count);

    //-----------Call your functions here--------------------
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    //--------------------------------------------------


    //---------------Correction check and speedup calculation----------------------
#if 0
    float speedup=0.0;
    if(checker()==1)
        speedup = time_overhead_default/time_overhead_student;
    cout<<"Speedup="<<speedup<<endl;
    //-----------------------------------------------------------------------------
#endif
    return 0;
}
